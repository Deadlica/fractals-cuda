#include "hip/hip_runtime.h"
#include <Fractal/mandelbrot.cuh>

__device__ Color linear_interpolate(const Color& color1, const Color& color2, double t) {
    unsigned char r = static_cast<unsigned char>(color1.r + t * (color2.r - color1.r));
    unsigned char g = static_cast<unsigned char>(color1.g + t * (color2.g - color1.g));
    unsigned char b = static_cast<unsigned char>(color1.b + t * (color2.b - color1.b));
    return Color(r, g, b);
}

__global__ void mandelbrot_kernel(Color* d_image, Color* PALETTE,
                                 int* palette_size, int width, int height,
                                 double x_min, double x_max, double y_min,
                                 double y_max, int max_iter,
                                 bool smooth) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= width || idy >= height) {
        return;
    }

    double x0 = x_min + idx * (x_max - x_min) / width;
    double y0 = y_min + idy * (y_max - y_min) / height;
    double x = 0.0, y = 0.0;
    int iter = 0;

    while (x * x + y * y <= 4.0 && iter < max_iter) {
        double xtemp = x * x - y * y + x0;
        y = 2 * x * y + y0;
        x = xtemp;
        iter++;
    }

    if (smooth) {
        double t;
        double iter_d;
        if (iter < max_iter) {
            double log_zn = logf(x * x + y * y) / 2.0f;
            double nu = logf(log_zn / logf(2.0f)) / logf(2.0f);
            iter_d = iter + 1 - nu;
            iter = static_cast<int>(floor(iter_d));
        }
        t = iter_d - iter;
        Color color1 = PALETTE[iter % *palette_size];
        Color color2 = PALETTE[(iter + 1) % *palette_size];
        Color color = linear_interpolate(color1, color2, t);
        d_image[idy * width + idx] = color;
    }
    else {
        d_image[idy * width + idx] = PALETTE[iter % *palette_size];
    }
}

void mandelbrot(Color* h_image, int width, int height, double x_min,
                double x_max, double y_min, double y_max, int max_iter, bool smooth) {
    Color* d_image;
    size_t image_size = width * height * sizeof(Color);
    hipMalloc(&d_image, image_size);

    dim3 block_size(32, 32);
    dim3 grid_size((width + block_size.x - 1) / block_size.x,
                  (height + block_size.y - 1) / block_size.y);

    mandelbrot_kernel<<<grid_size, block_size>>>(d_image, PALETTE, PALETTE_SIZE,
                                               width, height, x_min, x_max, y_min,
                                               y_max, max_iter, smooth);
    hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
    hipFree(d_image);
}
