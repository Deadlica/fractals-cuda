// Project
#include <Fractal/palette.cuh>
#include <Util/util.h>

// std
#include <vector>
#include <fstream>
#include <iostream>

// Default theme
std::vector<Color> v_palette = {
    Color(66, 30, 15),
    Color(25, 7, 26),
    Color(9, 1, 47),
    Color(4, 4, 73),
    Color(0, 7, 100),
    Color(12, 44, 138),
    Color(24, 82, 177),
    Color(57, 125, 209),
    Color(134, 181, 229),
    Color(211, 236, 248),
    Color(241, 233, 191),
    Color(248, 201, 95),
    Color(255, 170, 0),
    Color(204, 128, 0),
    Color(153, 87, 0),
    Color(106, 52, 3)
};

int v_palette_size = v_palette.size();


Color* PALETTE = nullptr;
int* PALETTE_SIZE = nullptr;

std::string get_theme_path(const std::string& theme) {
    std::string path = theme;
    if (!util::starts_with(theme, "themes/")) {
        path = "themes/" + theme;
    }
    if (!util::ends_with(theme, ".mbt")) {
        path += ".mbt";
    }

    return path;
}

void load_color_theme(const std::string& path) {
    std::ifstream file(path);
    if (!file.is_open()) {
        std::cout << "Could not find file: " << path << std::endl << "Using default." << std::endl;
        return;
    }

    std::vector<Color> temp_palette;
    
    std::string line;
    while (std::getline(file, line)) {
        size_t comma1 = line.find(",");
        if (comma1 == std::string::npos) {
            std::cout << "Invalid theme: " << path << std::endl << "Using default theme." << std::endl;
            file.close();
            return;
        }

        size_t comma2 = line.find(",", comma1 + 1);
        if (comma2 == std::string::npos) {
            std::cout << "Invalid theme: " << path << std::endl << "Using default theme." << std::endl;
            file.close();
            return;
        }

        try {
            int r = std::stoi(line.substr(0, comma1));
            int g = std::stoi(line.substr(comma1 + 1, comma2 - comma1 - 1));
            int b = std::stoi(line.substr(comma2 + 1));

            temp_palette.push_back(Color{static_cast<unsigned char>(r),
                                         static_cast<unsigned char>(g),
                                         static_cast<unsigned char>(b)});
        } catch (const std::invalid_argument& e) {
            std::cout << "Invalid value in theme: " << path << std::endl << "Using default theme." << std::endl;
            file.close();
            return;
        }
    }

    v_palette = temp_palette;
    v_palette_size = temp_palette.size();
}

void initialize_palette(const std::string& theme) {
    if (!theme.empty()) {
        std::string file_name = get_theme_path(theme);
        load_color_theme(file_name);
    }

    size_t palette_size = v_palette.size() * sizeof(Color);
    size_t int_size = sizeof(v_palette_size);

    hipMalloc(&PALETTE, palette_size);
    hipMalloc(&PALETTE_SIZE, int_size);

    hipMemcpy(PALETTE, v_palette.data(), palette_size, hipMemcpyHostToDevice);
    hipMemcpy(PALETTE_SIZE, &v_palette_size, int_size, hipMemcpyHostToDevice);
}

void free_palette() {
    hipFree(PALETTE);
    hipFree(PALETTE_SIZE);
}
